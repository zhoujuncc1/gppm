#include "../../include/Model.h"
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
GPUFileTrajectoryProvider::GPUFileTrajectoryProvider(std::string filename) : FileTrajectoryProvider(filename)
    {
        dim_array[TRAJ_N_SPECIES_I] = N_SPECIES;
        dim_array[TRAJ_END_TIME_I] = end_time;
        dim_array[TRAJ_SIZE_I] = size;

        trajectories_host = (float *)malloc(size * N_SPECIES * 2 * end_time);

        hipError_t error;
        error = hipMalloc((void **)&dim_array_device, 3 * sizeof(int));

        if (error != hipSuccess)
        {
            printf("hipMalloc dim_array_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        error = hipMemcpy(dim_array_device, dim_array, 3 * sizeof(float), hipMemcpyHostToDevice);

        if (error != hipSuccess)
        {
            printf("hipMemcpy (dim_array_device, dim_array) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&trajectories_device, 2 * N_SPECIES * end_time * size * sizeof(float));

        if (error != hipSuccess)
        {
            printf("hipMalloc trajectories_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        std::ifstream input(filename);
        input >> N_SPECIES;
        input >> end_time;
        input >> size;

        float *trajectory_host = (float *)malloc(2 * N_SPECIES * end_time * sizeof(float));
        hipError_t error;

        for (int k = 0; k < size; k++)
        {
            int index = 0;
            GPUTrajectory traj(N_SPECIES, end_time);
            error = hipMalloc((void **)&(traj.traj_device), 2 * N_SPECIES * end_time * sizeof(float));
            if (error != hipSuccess)
            {
                printf("hipMalloc traj.traj_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
            }

            {
                state_type state;
                for (int i = 0; i < N_SPECIES; i++)
                {
                    double v;
                    input >> v;
                    trajectory_host[index++] = v;
                    state.push_back(v);
                }
                for (int i = 0; i < N_SPECIES; i++)
                {
                    state.push_back(0);
                }

                traj.m_states.push_back(state);
                traj.m_times.push_back(0);
            }
            for (int t = 1; t < end_time; t++)
            {
                state_type state;
                double v;
                for (int i = 0; i < N_SPECIES; i++)
                {
                    input >> v;
                    trajectory_host[index++] = v;
                    state.push_back(v);
                }
                for (int i = 0; i < N_SPECIES; i++)
                {
                    state.push_back(state[i] - traj.m_states.back()[i]);
                }

                traj.m_states.push_back(state);
                traj.m_times.push_back(t);
            }

            error = hipMemcpy(traj.traj_device, trajectory_host, 2 * N_SPECIES * end_time * sizeof(float), hipMemcpyHostToDevice);

            if (error != hipSuccess)
            {
                printf("hipMemcpy (traj.traj_device, trajectory_host) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
                exit(EXIT_FAILURE);
            }
            trajectories.push_back(traj);
            memcpy(trajectories_host + k * 2 * N_SPECIES * end_time, trajectory_host, 2 * N_SPECIES * end_time * sizeof(float));
        }
        error = hipMemcpy(trajectories_device, trajectories_host, size * 2 * N_SPECIES * end_time * sizeof(float), hipMemcpyHostToDevice);

        if (error != hipSuccess)
        {
            printf("hipMemcpy (traj.traj_device, trajectory_host) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        input.close();
    }