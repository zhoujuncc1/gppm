#include "../../include/Model.h"
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
GPUFileTrajectoryProvider::GPUFileTrajectoryProvider(std::string filename)
    {

      printf("Enter GPU Traj Provider!\n");
        std::ifstream input(filename);
        input >> N_SPECIES;
        input >> end_time;
        input >> size;
    
        dim_array[TRAJ_N_SPECIES_I] = N_SPECIES;
        dim_array[TRAJ_END_TIME_I] = end_time;
        dim_array[TRAJ_SIZE_I] = size;


        hipError_t error;
        error = hipMalloc((void **)&dim_array_device, 3 * sizeof(int));
        if (error != hipSuccess)
        {
            printf("hipMalloc dim_array_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        error = hipMemcpy(dim_array_device, dim_array, 3 * sizeof(float), hipMemcpyHostToDevice);

        if (error != hipSuccess)
        {
            printf("hipMemcpy (dim_array_device, dim_array) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        trajectories_host = (float *)malloc(size * N_SPECIES * 2 * end_time*sizeof(float));
        error = hipMalloc((void **)&trajectories_device, 2 * N_SPECIES * end_time * size * sizeof(float));

        if (error != hipSuccess)
        {
            printf("hipMalloc trajectories_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        int index = 0;

        for (int k = 0; k < size; k++)
        {
            Trajectory traj(N_SPECIES, end_time);

            {
                state_type state;
                for (int i = 0; i < N_SPECIES; i++)
                {
                    float v;
                    input >> v;
                    trajectories_host[index++] = v;
                    state.push_back(v);
                }
                for (int i = 0; i < N_SPECIES; i++)
                {
                    state.push_back(0);
                    trajectories_host[index++] = 0;
                }

                traj.m_states.push_back(state);
                traj.m_times.push_back(0);
            }
            for (int t = 1; t < end_time; t++)
            {
                state_type state;
                for (int i = 0; i < N_SPECIES; i++)
                {
                    float v;
                    input >> v;
                    trajectories_host[index++] = v;
                    state.push_back(v);
                }
                for (int i = 0; i < N_SPECIES; i++)
                {
                    state.push_back(state[i] - traj.m_states.back()[i]);
                    trajectories_host[index++] = state[i] - traj.m_states.back()[i];
                }

                traj.m_states.push_back(state);
                traj.m_times.push_back(t);
            }
            trajectories.push_back(traj);

        }
        error = hipMemcpy(trajectories_device, trajectories_host, size * 2 * N_SPECIES * end_time * sizeof(float), hipMemcpyHostToDevice);

        if (error != hipSuccess)
        {
            printf("hipMemcpy (traj.trajectories_device, trajectories_host) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        input.close();
    }
