#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <vector>
#include <map>
#include <hip/hip_runtime.h>
#include "../../include/helper_functions.h"
#include "../../include/hip/hip_runtime_api.h"

#include "../../include/Bltl/BltlChecker.h"
#include "../../include/Bltl/Bltl.h"

__global__ void eval_prd_kernel(float *traj, bool *value, int varId, float left, float right, int *dim_array_device)
{
	int n = threadIdx.x;
    int t = threadIdx.y;
	int index = n*2*dim_array_device[TRAJ_N_SPECIES_I]*dim_array_device[TRAJ_END_TIME_I] +t*2*dim_array_device[TRAJ_N_SPECIES_I]+varId;
	value[t] = traj[index]<=right && traj[index]>=left;
}




__global__ void eval_G_kernel(bool* value_arr, bool* sub_value_arr, int distance, int *dim_array_device){
	int n = threadIdx.x;
    int t = threadIdx.y;
    int index = n*dim_array_device[TRAJ_END_TIME_I]+t;
    value_arr[t] = true;
	for(int k = 0 ; k < distance && t+k<dim_array_device[TRAJ_END_TIME_I]; k++){
	    value_arr[index] = value_arr[index] && sub_value_arr[index+k];
	}
}

__global__ void eval_F_kernel(bool* value_arr, bool* sub_value_arr, int distance, int *dim_array_device){
	int n = threadIdx.x;
    int t = threadIdx.y;
    int index = n*dim_array_device[TRAJ_END_TIME_I]+t;
	value_arr[t] = false;
	for(int k = 0 ; k < distance && t+k<dim_array_device[TRAJ_END_TIME_I]; k++){
	    value_arr[index] = value_arr[index] || sub_value_arr[index+k];
	}
}

void GPUBltlChecker::eval_prds()
{
	dim3 threadsPerBlock_prd(traj_provider->dim_array[TRAJ_SIZE_I], length);
	
	for (auto prd_itr = prds.begin(); prd_itr != prds.end(); prd_itr++)
	{
		Prd *prd = prd_itr->second;
		eval_prd_kernel<<<1, threadsPerBlock_prd>>>(traj_provider->trajectories_device, prd->value_array_device, prd->varId, prd->left->value, prd->right->value, dim_array_device);
	}
	hipDeviceSynchronize();
}

bool* GPUBltlChecker::eval_bltl_recursive(Bltl *bltl)
{
    if(bltl->getOperation() == op_PRD)
        return bltl->getPrd()->value_array_device;
	else
	{
        bool* sub_value_arr = eval_bltl_recursive(bltl->getChild1());
		hipError_t error = hipMalloc((void **)&(bltl->value_array_device), traj_provider->dim_array[TRAJ_SIZE_I]*length * sizeof(bool));
		if (error != hipSuccess)
		{
			printf("hipMalloc bltl->value_array_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
			exit(EXIT_FAILURE);
		}
		dim3 threadsPerBlock(traj_provider->dim_array[TRAJ_SIZE_I], length);
        if(bltl->getOperation() == op_G)
            eval_G_kernel<<<1, threadsPerBlock>>>(bltl->value_array_device, sub_value_arr, bltl->getTime()->value, dim_array_device);
        else if(bltl->getOperation() == op_F)
            eval_F_kernel<<<1, threadsPerBlock>>>(bltl->value_array_device, sub_value_arr, bltl->getTime()->value, dim_array_device);
        hipDeviceSynchronize();
        return bltl->value_array_device;
	}
}
vector<int> GPUBltlChecker::eval_bltl(Bltl *bltl){
    eval_bltl_recursive(bltl);
    bool* result_arr = (bool*) malloc(traj_provider->dim_array[TRAJ_SIZE_I]*length);

    hipError_t error = hipMemcpy(result_arr, bltl->value_array_device, traj_provider->dim_array[TRAJ_SIZE_I]*length * sizeof(bool), hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (traj.traj_device, trajectory_host) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    
    vector<int> results;
    for(int i = 0; i < traj_provider->dim_array[TRAJ_SIZE_I]; i++)
        results.push_back(result_arr[i*length]);
    return results;
}

GPUBltlChecker::GPUBltlChecker(Bltl *bltl, map<string, Prd *> prds, GPUFileTrajectoryProvider* traj_provider)
	{
        isGPU=true;
		this->bltl=bltl;
		this->prds=prds;
		this->traj_provider = traj_provider;
		hipError_t error;
		for (auto prd_itr = prds.begin(); prd_itr != prds.end(); prd_itr++)
		{
			Prd *prd = prd_itr->second;
			error = hipMalloc((void **)&(prd->value_array_device), traj_provider->dim_array[TRAJ_SIZE_I]*length * sizeof(bool));
			if (error != hipSuccess)
			{
				printf("hipMalloc prd->value_array_device returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
				exit(EXIT_FAILURE);
			}
		}
        dim_array_device = traj_provider->dim_array_device;
        
	}
