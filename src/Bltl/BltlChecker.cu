#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <map>
#include <hip/hip_runtime.h>
#include "../../include/helper_functions.h"
#include "../../include/hip/hip_runtime_api.h"

#include "../../include/Bltl/BltlChecker.h"
#include "../../include/Bltl/Bltl.h"

__global__ void fill_bool_array(bool* to_fill, bool value, int n_threads) {
	int n = blockIdx.x * n_threads + threadIdx.x;
	to_fill[n] = value;
}

__global__ void eval_prd_kernel(float *traj, bool *value_default_true,
		bool *value_default_false, int varId, float left, float right,
		int *dim_array_device) {
	int n = blockIdx.x;
	int t = threadIdx.x;
	int index = n * 2 * dim_array_device[TRAJ_N_SPECIES_I]
			* dim_array_device[TRAJ_END_TIME_I]
			+ t * 2 * dim_array_device[TRAJ_N_SPECIES_I] + varId;
	value_default_true[n * 2 * dim_array_device[TRAJ_END_TIME_I] + t] =
			traj[index] <= right && traj[index] >= left;
	value_default_false[n * 2 * dim_array_device[TRAJ_END_TIME_I] + t] =
			value_default_true[n * 2 * dim_array_device[TRAJ_END_TIME_I] + t];
}

__global__ void eval_G_kernel(bool* value_arr_default_true,
		bool* value_arr_default_false, bool* sub_value_arr, int distance,
		int *dim_array_device) {
	int n = blockIdx.x;
	int t = threadIdx.x;
	int index = n * 2 * dim_array_device[TRAJ_END_TIME_I] + t;
	value_arr_default_true[index] = true;
	for (int k = 0; k < distance; k++) {
		value_arr_default_true[index] = value_arr_default_true[index]
				&& sub_value_arr[index + k];
	}
	value_arr_default_false[index] = value_arr_default_true[index];
}

__global__ void eval_F_kernel(bool* value_arr_default_true,
		bool* value_arr_default_false, bool* sub_value_arr, int distance,
		int *dim_array_device) {
	int n = blockIdx.x;
	int t = threadIdx.x;
	int index = n * 2 * dim_array_device[TRAJ_END_TIME_I] + t;
	value_arr_default_false[index] = false;
	for (int k = 0; k < distance; k++) {
		value_arr_default_false[index] = value_arr_default_false[index]
				|| sub_value_arr[index + k];
	}
	value_arr_default_true[index] = value_arr_default_false[index];
}

__global__ void eval_AND_kernel(bool* value_arr_default_true,
		bool* value_arr_default_false, bool* sub_value_arr1,
		bool* sub_value_arr2, int *dim_array_device) {
	int n = blockIdx.x;
	int t = threadIdx.x;
	int index = n * 2 * dim_array_device[TRAJ_END_TIME_I] + t;
	value_arr_default_true[index] = sub_value_arr1[index]
			&& sub_value_arr2[index];
	value_arr_default_false[index] = value_arr_default_true[index];

}

__global__ void eval_OR_kernel(bool* value_arr_default_true,
		bool* value_arr_default_false, bool* sub_value_arr1,
		bool* sub_value_arr2, int *dim_array_device) {
	int n = blockIdx.x;
	int t = threadIdx.x;
	int index = n * 2 * dim_array_device[TRAJ_END_TIME_I] + t;
	value_arr_default_true[index] = sub_value_arr1[index]
			|| sub_value_arr2[index];
	value_arr_default_false[index] = value_arr_default_true[index];

}
void GPUBltlChecker::eval_prds() {
	dim3 block_number_prd(traj_provider->dim_array[TRAJ_SIZE_I]);
	dim3 threadsPerBlock_prd(length);
	for (auto prd_itr = prds.begin(); prd_itr != prds.end(); prd_itr++) {
		Prd *prd = prd_itr->second;
		eval_prd_kernel<<<block_number_prd, threadsPerBlock_prd>>>(
				traj_provider->trajectories_device,
				prd->value_array_device_default_true,
				prd->value_array_device_default_false, prd->varId,
				prd->left->value, prd->right->value, dim_array_device);
		hipDeviceSynchronize();

	}
}

void GPUBltlChecker::eval_prd(Prd* prd) {
	dim3 block_number_prd(traj_provider->dim_array[TRAJ_SIZE_I]);
	dim3 threadsPerBlock_prd(length);
	eval_prd_kernel<<<block_number_prd, threadsPerBlock_prd>>>(
			traj_provider->trajectories_device,
			prd->value_array_device_default_true,
			prd->value_array_device_default_false, prd->varId, prd->left->value,
			prd->right->value, dim_array_device);
	hipDeviceSynchronize();
}

bool* GPUBltlChecker::eval_bltl_recursive(Bltl *bltl) {
	if (bltl->getOperation() == op_PRD) {
		eval_prd(bltl->getPrd());
	}
	else {
		if (bltl->getChild1())
			eval_bltl_recursive(bltl->getChild1());
		if (bltl->getChild2())
			eval_bltl_recursive(bltl->getChild2());

		dim3 block_number_prd(traj_provider->dim_array[TRAJ_SIZE_I]);
		dim3 threadsPerBlock_prd(length);

		if (bltl->getOperation() == op_G)
			eval_G_kernel<<<block_number_prd, threadsPerBlock_prd>>>(
					bltl->value_array_device_default_true,
					bltl->value_array_device_default_false,
					bltl->getChild1()->value_array_device_default_true,
					bltl->getTime()->value, dim_array_device);
		else if (bltl->getOperation() == op_F)
			eval_F_kernel<<<block_number_prd, threadsPerBlock_prd>>>(
					bltl->value_array_device_default_true,
					bltl->value_array_device_default_false,
					bltl->getChild1()->value_array_device_default_false,
					bltl->getTime()->value, dim_array_device);
		else if (bltl->getOperation() == op_AND)
			eval_AND_kernel<<<block_number_prd, threadsPerBlock_prd>>>(
					bltl->value_array_device_default_true,
					bltl->value_array_device_default_false,
					bltl->getChild1()->value_array_device_default_true,
					bltl->getChild2()->value_array_device_default_true,
					dim_array_device);
		else if (bltl->getOperation() == op_OR)
			eval_OR_kernel<<<block_number_prd, threadsPerBlock_prd>>>(
					bltl->value_array_device_default_true,
					bltl->value_array_device_default_false,
					bltl->getChild1()->value_array_device_default_true,
					bltl->getChild2()->value_array_device_default_true,
					dim_array_device);

		hipDeviceSynchronize();
	}
}

vector<int> GPUBltlChecker::eval_bltl(Bltl *bltl) {
	eval_bltl_recursive(bltl);

	hipError_t error = hipMemcpy(result_arr,
			bltl->value_array_device_default_true,
			2 * traj_provider->dim_array[TRAJ_SIZE_I] * length * sizeof(bool),
			hipMemcpyDeviceToHost);

	if (error != hipSuccess) {
		printf(
				"hipMemcpy (result_arr, bltl->value_array_device) returned error %s (code %d), line(%d)\n",
				hipGetErrorString(error), error, __LINE__);
		exit(EXIT_FAILURE);
	}

	vector<int> results;
	for (int i = 0; i < traj_provider->dim_array[TRAJ_SIZE_I]; i++)
		results.push_back(result_arr[i * 2 * length]);
	return results;
}

void GPUBltlChecker::do_init_array(Bltl* bltl) {
	hipMalloc((void **) &(bltl->value_array_device_default_true),
			2 * traj_provider->dim_array[TRAJ_SIZE_I] * length * sizeof(bool));

	hipMemset(bltl->value_array_device_default_true, true,
			2 * traj_provider->dim_array[TRAJ_SIZE_I] * length * sizeof(bool));

	hipMalloc((void **) &(bltl->value_array_device_default_false),
			2 * traj_provider->dim_array[TRAJ_SIZE_I] * length * sizeof(bool));

	hipMemset(bltl->value_array_device_default_false, false,
			2 * traj_provider->dim_array[TRAJ_SIZE_I] * length * sizeof(bool));
}

void GPUBltlChecker::init_value_array(Bltl* bltl) {
	do_init_array(bltl);
	if (bltl->getOperation() == op_PRD) {
		bltl->getPrd()->value_array_device_default_true =
				bltl->value_array_device_default_true;
		bltl->getPrd()->value_array_device_default_false =
				bltl->value_array_device_default_false;
	} else {
		if (bltl->getChild1())
			init_value_array(bltl->getChild1());
		if (bltl->getChild2())
			init_value_array(bltl->getChild2());
	}
}

vector<int> GPUBltlChecker::check() {
	//eval_prds();
	vector<int> result = eval_bltl(bltl);
	return result;
}

GPUBltlChecker::GPUBltlChecker(Bltl *bltl, map<string, Prd *> prds,
		GPUFileTrajectoryProvider* traj_provider) {
	isGPU = true;
	this->bltl = bltl;
	this->prds = prds;
	this->traj_provider = traj_provider;
	this->length = traj_provider->dim_array[TRAJ_END_TIME_I];
	dim_array_device = traj_provider->dim_array_device;
	init_value_array(this->bltl);
	result_arr = (bool*) malloc(
			2 * traj_provider->dim_array[TRAJ_SIZE_I] * length * sizeof(bool));
}
